#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "simulate.h"

#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#else
#define EXPORT
#endif

// **Helperfuncties voor gebruik op de GPU**
__device__ int determineWinner(char move1, char move2) {
    int move1Idx = -1;
    if (move1 == 'W') move1Idx = 0;
    else if (move1 == 'V') move1Idx = 1;
    else if (move1 == 'A') move1Idx = 2;
    else if (move1 == 'L') move1Idx = 3;
    else if (move1 == 'D') move1Idx = 4;

    int move2Idx = -1;
    if (move2 == 'W') move2Idx = 0;
    else if (move2 == 'V') move2Idx = 1;
    else if (move2 == 'A') move2Idx = 2;
    else if (move2 == 'L') move2Idx = 3;
    else if (move2 == 'D') move2Idx = 4;

    if (move1Idx == -1 || move2Idx == -1) return 0;

    const int moveWins[5][5] = {
        {0, 1, 0, 2, 0}, // W
        {2, 0, 1, 0, 0}, // V
        {0, 2, 0, 1, 0}, // A
        {1, 0, 2, 0, 0}, // L
        {0, 0, 0, 0, 0}  // D
    };
    return moveWins[move1Idx][move2Idx];
}

__device__ int getIndex(char move) {
    switch (move) {
        case 'W': return 0;
        case 'V': return 1;
        case 'A': return 2;
        case 'L': return 3;
        case 'D': return 4;
        default: return -1;
    }
}

__device__ char getElementFromCode(int depth) {
    if (depth < 1 || depth > 5) return 0;
    const char depthToElement[5] = {'W', 'V', 'A', 'L', 'D'};
    return depthToElement[depth - 1];
}

__device__ char getElementByDepth(char prevElement, int depth) {
    if (depth == 5) return 'D';
    if (prevElement == 0) return 0;
    if (prevElement == 'D') prevElement = 'L';
    if (depth < 1 || depth > 4) return 0;
    const char elementsDepth[4][4] = {
        {'L', 'A', 'V', 'W'}, // W
        {'W', 'L', 'A', 'V'}, // V
        {'V', 'W', 'L', 'A'}, // A
        {'A', 'V', 'W', 'L'}  // L
    };
    int idx = -1;
    if (prevElement == 'W') idx = 0;
    else if (prevElement == 'V') idx = 1;
    else if (prevElement == 'A') idx = 2;
    else if (prevElement == 'L') idx = 3;
    if (idx == -1) return 0;
    return elementsDepth[idx][depth - 1];
}

__device__ char chooseAvailableElement(char target, int* available) {
    int targetIdx = getIndex(target);
    if (targetIdx != -1 && available[targetIdx] > 0) {
        return target;
    }
    char current = target;
    for (int i = 0; i < 5; i++) {
        if (current == 'W') current = 'L';
        else if (current == 'V') current = 'W';
        else if (current == 'A') current = 'V';
        else if (current == 'L') current = 'A';
        else break;
        int currentIdx = getIndex(current);
        if (currentIdx != -1 && available[currentIdx] > 0) {
            return current;
        }
    }
    if (available[4] > 0) { // D
        return 'D';
    }
    return 0;
}

__device__ char getLastElement(int* available) {
    const char candidates[5] = {'W', 'V', 'A', 'L', 'D'};
    for (int i = 0; i < 5; i++) {
        char c = candidates[i];
        int idx = getIndex(c);
        if (available[idx] > 0) {
            return c;
        }
    }
    return 0;
}

// **CUDA Kernels**
__global__ void simulateDepthVsFixedKernel(const char* generatedEngines, const char* fixedInputEngines,
                                           int numGenerated, int numFixed, int* scoreDiffs) {
    int generatedIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int fixedIdx = blockIdx.y * blockDim.y + threadIdx.y;
    if (generatedIdx < numGenerated && fixedIdx < numFixed) {
        const char* engine = generatedEngines + generatedIdx * 12;
        const char* fixedEngine = fixedInputEngines + fixedIdx * 13;
        
        int available[5] = {3, 3, 3, 3, 1}; // W, V, A, L, D
        char moves[13];
        
        for (int i = 0; i < 12; i++) {
            int depth = engine[i] - '0';
            char prevMove = (i == 0) ? 0 : fixedEngine[i-1];
            char target = (i == 0) ? getElementFromCode(depth) : getElementByDepth(prevMove, depth);
            char move = chooseAvailableElement(target, available);
            if (move == 0) {
                move = 'W'; // Default
            }
            int moveIdx = getIndex(move);
            available[moveIdx]--;
            moves[i] = move;
        }
        
        char lastMove = getLastElement(available);
        if (lastMove != 0) {
            int lastIdx = getIndex(lastMove);
            available[lastIdx]--;
            moves[12] = lastMove;
        } else {
            moves[12] = 'W';
        }
        
        int p1Score = 0, p2Score = 0;
        for (int i = 0; i < 13; i++) {
            char move1 = moves[i];
            char move2 = fixedEngine[i];
            int winner = determineWinner(move1, move2);
            if (winner == 1) p1Score++;
            else if (winner == 2) p2Score++;
        }
        
        // Nieuwe scoreberekening
        int diff = p1Score - p2Score;
        if (p1Score > p2Score) {
            scoreDiffs[generatedIdx * numFixed + fixedIdx] = diff + 10; // Winst: +10
        } else if (p1Score < p2Score) {
            scoreDiffs[generatedIdx * numFixed + fixedIdx] = diff - 10; // Verlies: -10
        } else {
            scoreDiffs[generatedIdx * numFixed + fixedIdx] = p1Score;   // Gelijkspel: p1Score
        }
    }
}

// Kernel voor Depth vs Depth
__global__ void simulateDepthVsDepthKernel(const char* generatedEngines, const char* depthInputEngines,
                                           int numGenerated, int numDepth, int* scoreDiffs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < numGenerated && idy < numDepth) {
        const char* engine1 = generatedEngines + idx * 12;
        const char* engine2 = depthInputEngines + idy * 12;
        
        int available1[5] = {3, 3, 3, 3, 1};
        int available2[5] = {3, 3, 3, 3, 1};
        char moves1[13], moves2[13];
        int p1Score = 0, p2Score = 0;
        
        for (int i = 0; i < 12; i++) {
            int depth1 = engine1[i] - '0';
            int depth2 = engine2[i] - '0';
            char prevMove2 = (i == 0) ? 0 : moves2[i-1];
            char prevMove1 = (i == 0) ? 0 : moves1[i-1];
            
            char target1 = (i == 0) ? getElementFromCode(depth1) : getElementByDepth(prevMove2, depth1);
            char target2 = (i == 0) ? getElementFromCode(depth2) : getElementByDepth(prevMove1, depth2);
            
            char move1 = chooseAvailableElement(target1, available1);
            if (move1 == 0) move1 = 'W';
            char move2 = chooseAvailableElement(target2, available2);
            if (move2 == 0) move2 = 'W';
            
            available1[getIndex(move1)]--;
            available2[getIndex(move2)]--;
            moves1[i] = move1;
            moves2[i] = move2;
            
            int winner = determineWinner(move1, move2);
            if (winner == 1) p1Score++;
            else if (winner == 2) p2Score++;
        }
        
        char lastMove1 = getLastElement(available1);
        if (lastMove1 != 0) available1[getIndex(lastMove1)]--;
        else lastMove1 = 'W';
        char lastMove2 = getLastElement(available2);
        if (lastMove2 != 0) available2[getIndex(lastMove2)]--;
        else lastMove2 = 'W';
        moves1[12] = lastMove1;
        moves2[12] = lastMove2;
        
        int winner = determineWinner(lastMove1, lastMove2);
        if (winner == 1) p1Score++;
        else if (winner == 2) p2Score++;
        
        // Nieuwe scoreberekening
        int diff = p1Score - p2Score;
        if (p1Score > p2Score) {
            scoreDiffs[idx * numDepth + idy] = diff + 10; // Winst: +10
        } else if (p1Score < p2Score) {
            scoreDiffs[idx * numDepth + idy] = diff - 10; // Verlies: -10
        } else {
            scoreDiffs[idx * numDepth + idy] = p1Score;   // Gelijkspel: p1Score
        }
    }
}

// **Wrapper-functies voor cgo**
extern "C" void simulateDepthVsFixedCUDA(const char* generatedEngines, int numGenerated,
                                         const char* fixedInputEngines, int numFixed, int* scoreDiffs) {
    char *d_generatedEngines, *d_fixedInputEngines;
    int *d_scoreDiffs;

    // Alloceer geheugen op de GPU met de juiste grootte
    hipMalloc(&d_generatedEngines, numGenerated * 12 * sizeof(char));  // 12 bytes per generated engine
    hipMalloc(&d_fixedInputEngines, numFixed * 13 * sizeof(char));     // 13 bytes per fixed engine
    hipMalloc(&d_scoreDiffs, numGenerated * numFixed * sizeof(int));

    // Kopieer data van host naar GPU met de juiste grootte
    hipMemcpy(d_generatedEngines, generatedEngines, numGenerated * 12 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_fixedInputEngines, fixedInputEngines, numFixed * 13 * sizeof(char), hipMemcpyHostToDevice);

    // Definieer block- en griddimensies
    dim3 blockDim(32, 32);
    dim3 gridDim((numGenerated + blockDim.x - 1) / blockDim.x, (numFixed + blockDim.y - 1) / blockDim.y);

    // Start de kernel
    simulateDepthVsFixedKernel<<<gridDim, blockDim>>>(d_generatedEngines, d_fixedInputEngines, numGenerated, numFixed, d_scoreDiffs);

    // Kopieer resultaten terug naar host
    hipMemcpy(scoreDiffs, d_scoreDiffs, numGenerated * numFixed * sizeof(int), hipMemcpyDeviceToHost);

    // Vrij GPU-geheugen
    hipFree(d_generatedEngines);
    hipFree(d_fixedInputEngines);
    hipFree(d_scoreDiffs);
}

// Wrapper voor Depth vs Depth
extern "C" SIMULATE_API void simulateDepthVsDepthCUDA(const char* generatedEngines, int numGenerated,
                                                      const char* depthInputEngines, int numDepth, int* scoreDiffs) {
    char *d_generatedEngines, *d_depthInputEngines;
    int *d_scoreDiffs;
    hipMalloc(&d_generatedEngines, numGenerated * 12 * sizeof(char));
    hipMalloc(&d_depthInputEngines, numDepth * 12 * sizeof(char));
    hipMalloc(&d_scoreDiffs, numGenerated * numDepth * sizeof(int));
    hipMemcpy(d_generatedEngines, generatedEngines, numGenerated * 12 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_depthInputEngines, depthInputEngines, numDepth * 12 * sizeof(char), hipMemcpyHostToDevice);
    dim3 blockDim(32, 32);
    dim3 gridDim((numGenerated + blockDim.x - 1) / blockDim.x, (numDepth + blockDim.y - 1) / blockDim.y);
    simulateDepthVsDepthKernel<<<gridDim, blockDim>>>(d_generatedEngines, d_depthInputEngines, numGenerated, numDepth, d_scoreDiffs);
    hipMemcpy(scoreDiffs, d_scoreDiffs, numGenerated * numDepth * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_generatedEngines);
    hipFree(d_depthInputEngines);
    hipFree(d_scoreDiffs);
}
